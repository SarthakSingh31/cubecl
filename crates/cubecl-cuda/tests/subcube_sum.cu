
#include <hip/hip_runtime.h>
typedef unsigned char uint8;
typedef unsigned short uint16;
typedef unsigned int uint;
typedef unsigned long long int uint64;
typedef long long int int64;

extern "C" __global__ void kernel(float output_0[], uint info[]) {

  int threadIdxGlobal = threadIdx.x + threadIdx.y * blockDim.x +
                        threadIdx.z * (blockDim.x * blockDim.y);

  int warpSizeChecked = min(warpSize, blockDim.x * blockDim.y * blockDim.z);
  float l_0_0;
  float l_0_1;
  bool l_0_2;
  uint l_0_3;
  l_0_3 = info[uint(0)];
  l_0_0 = (threadIdxGlobal < l_0_3) ? output_0[threadIdxGlobal] : float(0);

  l_0_1 = l_0_0;
  {
    for (int offset = 1; offset < warpSizeChecked; offset *= 2) {
      l_0_1 += __shfl_xor_sync(-1, l_0_1, offset);
    }
  }
  l_0_2 = threadIdxGlobal == uint(0);
  if (l_0_2) {
    uint l_1_0;
    bool l_1_1;
    l_1_0 = info[uint(0)];
    l_1_1 = uint(0) < l_1_0;
    if (l_1_1) {
      output_0[uint(0)] = l_0_1;
    }
  }
}
